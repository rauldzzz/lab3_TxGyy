#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define N 1024
#define ROWSIZE 9
#define THREADS_PER_BLOCK 128


__global__ void cuspmv(int m, int r, double* dvals, int *dcols, double* dx, double *dy)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        double y_temp = 0.0;
        for (int j = 0; j < r; j++) {
            y_temp += dvals[j + row * r] * dx[dcols[j + row * r]];
        }
        dy[row] = y_temp;
    }

}


void spmv_cpu(int m, int r, double* vals, int* cols, double* x, double* y)
{
    for (int i = 0; i < m; i++)
    {
        double y_temp = 0.0;
        for (int j = 0; j < r; j++)
        {
            y_temp += vals[j + i*r]*x[cols[j + i*r]];
        }
        y[i] = y_temp;
    }  
}


void fill_matrix(double* vals, int* cols)
{

    int indx[ROWSIZE];
    int row_count = 0;
    for(int j = 0; j < N ; j++){
        for(int i = 0; i < N; i++){

            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +  j     *N;
            indx[3] = i - 1 +  j     *N;
            indx[4] = i     +  j     *N;
            indx[5] = i + 1 +  j     *N;
            indx[6] = i + 2 +  j     *N;
            indx[7] = i     + (j + 1)*N;
            indx[8] = i     + (j + 2)*N;

            for(int row = 0; row < ROWSIZE; row++)
            {
                if(indx[row] < 0 || indx[row] >= N*N)
                {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                }
                else
                {
                    cols[row + row_count*ROWSIZE] = indx[row];
                    if(row == 4)
                    {
                        vals[row + row_count*ROWSIZE] = 0.95;
                    }
                    else
                    {
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }

    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main()
{
    int vec_size = N*N;

    float time_cpu, time_gpu;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* x     = (double*) malloc (vec_size*sizeof(double));
    double* y_cpu = (double*) malloc (vec_size*sizeof(double));
    double* y_gpu = (double*) malloc (vec_size*sizeof(double));

    double* Avals = (double*) malloc (ROWSIZE*vec_size*sizeof(double));
    int*    Acols = (int   *) malloc (ROWSIZE*vec_size*sizeof(int));


    // fill vector with sinusoidal for testing the code
    for(int i = 0; i < vec_size; i++)
    {
        x[i] = sin(i*0.01);
        y_cpu[i] = 0.0;
    }

    fill_matrix(Avals, Acols);

    // measure time of CPU implementation
    hipEventRecord(start);

    for (int i = 0; i < 100; ++i)
        spmv_cpu(vec_size, ROWSIZE, Avals, Acols, x, y_cpu);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_cpu, start, stop);


    // complete here your cuda code
    double* dx;
    double* dy_gpu;

    double* dAvals;
    int*    dAcols;


    // allocate arrays in GPU
    hipMalloc(&dx, vec_size * sizeof(double));
    hipMalloc(&dy_gpu, vec_size * sizeof(double));
    hipMalloc(&dAvals, ROWSIZE * vec_size * sizeof(double));
    hipMalloc(&dAcols, ROWSIZE * vec_size * sizeof(int));

    // transfer data to GPU
    hipMemcpy(dx, x, vec_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAvals, Avals, ROWSIZE * vec_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAcols, Acols, ROWSIZE * vec_size * sizeof(int), hipMemcpyHostToDevice);

    // calculate threads and blocks
    int blocks = (vec_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // create the gridBlock
    hipEventRecord(start);

    for( int i=0; i<100; i++){
        // call your GPU kernel here
        cuspmv<<<blocks, THREADS_PER_BLOCK>>>(vec_size, ROWSIZE, dAvals, dAcols, dx, dy_gpu);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);

    // transfer result to CPU RAM
    hipMemcpy(y_gpu, dy_gpu, vec_size * sizeof(double), hipMemcpyDeviceToHost);
    
    // free arrays in GPU
    hipFree(dx);
    hipFree(dy_gpu);
    hipFree(dAvals);
    hipFree(dAcols);

    // comparison between gpu and cpu results
    double norm2 = 0.0;
    for(int i = 0; i < vec_size; i++)
        norm2 += (y_cpu[i] - y_gpu[i])*(y_cpu[i] - y_gpu[i]);

    norm2 = sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, size %d\n", norm2, vec_size);

    printf("CPU Time: %lf\n", time_cpu/1000);
    printf("GPU Time: %lf\n", time_gpu/1000);

    // free CPU arrays
    free(x);
    free(y_cpu);
    free(y_gpu);
    free(Acols);
    free(Avals);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
